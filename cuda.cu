#include "hip/hip_runtime.h"
%%writefile example.cu
#include <stdio.h>
#include <stdlib.h>
#include <gd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define WIDTH 2000
#define HEIGHT 2000
#define NUM_ITER 100

__global__ void game_of_life_kernel(int *old_state, int *new_state, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int idx = y * width + x;
    int num_neighbors = 0;

    // Check the neighbors
    for (int i = -1; i <= 1; i++) {
        for (int j = -1; j <= 1; j++) {
            int nx = (x + i + width) % width;
            int ny = (y + j + height) % height;
            int nidx = ny * width + nx;
            if (old_state[nidx]) {
                num_neighbors++;
            }
        }
    }

    // Apply the rules of the Game of Life
    if (old_state[idx]) {
        if (num_neighbors == 2 || num_neighbors == 3) {
            new_state[idx] = 1;
        } else {
            new_state[idx] = 0;
        }
    } else {
        if (num_neighbors == 3) {
            new_state[idx] = 1;
        } else {
            new_state[idx] = 0;
        }
    }
}

int main() {
    //printf("slm");
    clock_t start_time, end_time;
    double elapsed_time;

    // Record start time
    start_time = clock();

    // Initialize the grid with some cells
    int grid[HEIGHT][WIDTH] = {{0}};
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            if (rand() % 2 == 0) {
                grid[i][j] = 1;
            }
        }
    }

    // Copy the grid to the device
    int *d_old_state, *d_new_state;
    hipMalloc(&d_old_state, WIDTH * HEIGHT * sizeof(int));
    hipMalloc(&d_new_state, WIDTH * HEIGHT * sizeof(int));
    hipMemcpy(d_old_state, grid, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);

    // Set up the block and grid dimensions
    dim3 block_dim(32, 32);
    dim3 grid_dim((WIDTH + block_dim.x - 1) / block_dim.x, (HEIGHT + block_dim.y - 1) / block_dim.y);

    // Run the simulation for a fixed number of iterations
    for (int i = 0; i < NUM_ITER; i++) {
        game_of_life_kernel<<<grid_dim, block_dim>>>(d_old_state, d_new_state, WIDTH, HEIGHT);
        hipDeviceSynchronize();

        // Swap the old and new states
        int *tmp = d_old_state;
        d_old_state = d_new_state;
        d_new_state = tmp;

        // Generate an image of the current state
        int *h_old_state = (int *)malloc(WIDTH * HEIGHT * sizeof(int));
        hipMemcpy(h_old_state, d_old_state, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);

        gdImagePtr im;
        FILE *fp;
        int black, white;

        im = gdImageCreate(WIDTH, HEIGHT);
        white = gdImageColorAllocate(im, 255, 255, 255);
        black = gdImageColorAllocate(im, 0, 0, 0);

        for (int i = 0; i <WIDTH; i++) {
for (int j = 0; j < HEIGHT; j++) {
if (h_old_state[j * WIDTH + i]) {
gdImageSetPixel(im, i, j, black);
} else {
gdImageSetPixel(im, i, j, white);
}
}
}
    char filename[256];
    sprintf(filename, "frame%d.png", i);
    fp = fopen(filename, "wb");
    gdImagePng(im, fp);
    fclose(fp);
    gdImageDestroy(im);

    free(h_old_state);
}

// Free memory
hipFree(d_old_state);
hipFree(d_new_state);

// Record end time
    end_time = clock();

    // Calculate elapsed time
    elapsed_time = ((double) (end_time - start_time)) / CLOCKS_PER_SEC;

    // Print elapsed time
    printf("Elapsed time: %f seconds\n", elapsed_time);

return 0;
}